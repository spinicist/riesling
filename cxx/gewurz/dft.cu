#include "hip/hip_runtime.h"
#include "dft.cuh"

#include "rl/log.hpp"

#define LIBCUDACXX_ENABLE_SIMPLIFIED_COMPLEX_OPERATIONS
#include <cub/device/device_for.cuh>
#include <cuda/experimental/stream.cuh>
#include <hip/hip_math_constants.h>

#include "types.cuh"

namespace cudax = cuda::experimental;

namespace gw::DFT {

void ThreeD::forward(DTensor<CuCxF, 3>::Span imgs, DTensor<CuCxF, 2>::Span ks) const
{
  rl::Log::Print("DFT", "Forward DFT");
  auto const  start = rl::Log::Now();
  int const   nS = ks.extent(0);
  int const   nT = ks.extent(1);
  int const   nST = nS * nT;
  int const   nI = imgs.extent(0);
  int const   nJ = imgs.extent(1);
  int const   nK = imgs.extent(2);
  int const   nIJK = nI * nJ * nK;
  float const scale = 1.f / std::sqrt(nIJK);

  thrust::for_each_n(thrust::cuda::par, thrust::make_counting_iterator(0), nST,
                     [imgs, traj = this->traj, ks, scale] __device__(int st) {
                       CuReal const pi2 = CuReal(2.f * HIP_PI_F);

                       int const    nS = ks.extent(0);
                       int const    nT = ks.extent(1);
                       int const    it = st / nS;
                       int const    is = st % nS;
                       float3 const k = traj(is, it);
                       ks(is, it) = 0.f;

                       int const nI = imgs.extent(0);
                       int const nJ = imgs.extent(1);
                       int const nK = imgs.extent(2);
                       for (int ik = 0; ik < nK; ik++) {
                         float const z = (ik - nK / 2.f) / (float)nK;
                         for (int ij = 0; ij < nJ; ij++) {
                           float const y = (ij - nJ / 2.f) / (float)nJ;
                           for (int ii = 0; ii < nI; ii++) {
                             float const x = (ii - nI / 2.f) / (float)nI;
                             float const p = pi2 * (k.x * x + k.y * y + k.z * z);
                             CuCxF const ep(scale * cuda::std::cos(-p), scale * cuda::std::sin(-p));
                             ks(is, it) += ep * imgs(ii, ij, ik);
                           }
                         }
                       }
                     });
  rl::Log::Print("DFT", "Forward DFT finished in {}", rl::Log::ToNow(start));
}

void ThreeD::adjoint(DTensor<CuCxF, 2>::Span ks, DTensor<CuCxF, 3>::Span imgs) const
{
  int const   nI = imgs.extent(0);
  int const   nJ = imgs.extent(1);
  int const   nK = imgs.extent(2);
  int const   nIJ = nI * nJ;
  int const   nIJK = nIJ * nK;
  float const scale = 1.f / cuda::std::sqrt(nIJK);

  rl::Log::Print("DFT", "Adjoint DFT {} {} -> {} {} {} T {} {} {}", ks.extent(0), ks.extent(1), nI, nJ, nK, traj.extent(0),
                 traj.extent(1), traj.extent(2));
  auto const start = rl::Log::Now();

  auto it = thrust::make_counting_iterator(0);
  thrust::for_each_n(thrust::cuda::par, it, nIJK, [ks, traj = this->traj, imgs, scale] __device__(int ijk) {
    CuReal const pi2 = CuReal(2.f * HIP_PI_F);
    int const    nI = imgs.extent(0);
    int const    nJ = imgs.extent(1);
    int const    nK = imgs.extent(2);
    int const    nIJ = nI * nJ;
    int const    nS = ks.extent(0);
    int const    nT = ks.extent(1);

    int const ik = ijk / nIJ;
    int const ij = ijk % nIJ / nI;
    int const ii = ijk % nIJ % nI;

    imgs(ii, ij, ik) = 0.f;

    float3 const r{(ii - nI / 2.f) / (float)nI, (ij - nJ / 2.f) / (float)nJ, (ik - nK / 2.f) / (float)nK};

    for (int it = 0; it < nT; it++) {
      for (int is = 0; is < nS; is++) {
        float3 const k = traj(is, it);
        float const  p = pi2 * (r.x * k.x + r.y * k.y + r.z * k.z);
        CuCxF const  ep(scale * cuda::std::cos(p), scale * cuda::std::sin(p));
        imgs(ii, ij, ik) += ep * ks(is, it);
      }
    }
  });
  rl::Log::Print("DFT", "Adjoint DFT finished in {}", rl::Log::ToNow(start));
}

template <int NP>
void ThreeDPacked<NP>::forward(DTensor<CuCxF, 4>::Span imgs, DTensor<CuCxF, 3>::Span ks) const
{
  if (NP != imgs.extent(0) || NP != ks.extent(0)) {
    throw rl::Log::Failure("DFT", "Packing dimension size mismatch");
  }
  rl::Log::Print("DFT", "Forward Packed DFT");
  auto const  start = rl::Log::Now();
  int const   nS = ks.extent(1);
  int const   nT = ks.extent(2);
  int const   nST = nS * nT;
  int const   nI = imgs.extent(1);
  int const   nJ = imgs.extent(2);
  int const   nK = imgs.extent(3);
  int const   nIJK = nI * nJ * nK;
  float const scale = 1.f / cuda::std::sqrt(nIJK);
  auto        it = thrust::make_counting_iterator(0);
  thrust::for_each_n(thrust::cuda::par, it, nST, [imgs, traj = this->traj, ks, scale] __device__(int st) {
    CuReal const pi2 = CuReal(2.f * HIP_PI_F);
    int const    nC = ks.extent(0);
    int const    nS = ks.extent(1);
    int const    nT = ks.extent(2);
    int const    it = st / nS;
    int const    is = st % nS;
    float3 const k = traj(is, it);

    int const nI = imgs.extent(1);
    int const nJ = imgs.extent(2);
    int const nK = imgs.extent(3);
    CuCxF temp[NP] = {CuCxF(0.), };
    for (int ik = 0; ik < nK; ik++) {
      float const z = (ik - nK / 2.f) / (float)nK;
      for (int ij = 0; ij < nJ; ij++) {
        float const y = (ij - nJ / 2.f) / (float)nJ;
        for (int ii = 0; ii < nI; ii++) {
          float const x = (ii - nI / 2.f) / (float)nI;
          float const p = pi2 * (k.x * x + k.y * y + k.z * z);
          CuCxF const ep(scale * cuda::std::cos(-p), scale * cuda::std::sin(-p));
          for (int ic = 0; ic < NP; ic++) {
            temp[ic] += ep * imgs(ic, ii, ij, ik);
          }
        }
      }
    }
    for (int ic = 0; ic < NP; ic++) {
      ks(ic, is, it) = temp[ic];
    }
  });
  rl::Log::Print("DFT", "Forward Packed DFT finished in {}", rl::Log::ToNow(start));
}

template<int NP>
void ThreeDPacked<NP>::adjoint(DTensor<CuCxF, 3>::Span ks, DTensor<CuCxF, 4>::Span imgs) const
{
  if (NP != imgs.extent(0) || NP != ks.extent(0)) {
    throw rl::Log::Failure("DFT", "Packing dimension size mismatch");
  }
  int const   nI = imgs.extent(1);
  int const   nJ = imgs.extent(2);
  int const   nK = imgs.extent(3);
  int const   nIJ = nI * nJ;
  int const   nIJK = nIJ * nK;
  float const scale = 1.f / cuda::std::sqrt(nIJK);
  rl::Log::Print("DFT", "Adjoint Packed DFT {} {} {} -> {} {} {} {} T {} {} {}", ks.extent(0), ks.extent(1), ks.extent(2),
                 imgs.extent(0), nI, nJ, nK, traj.extent(0), traj.extent(1), traj.extent(2));
  auto const start = rl::Log::Now();

  auto it = thrust::make_counting_iterator(0);
  thrust::for_each_n(thrust::cuda::par, it, nIJK, [ks, traj = this->traj, imgs, scale] __device__(int ijk) {
    CuReal const pi2 = CuReal(2.f * HIP_PI_F);
    int const    nC = imgs.extent(0);
    int const    nI = imgs.extent(1);
    int const    nJ = imgs.extent(2);
    int const    nK = imgs.extent(3);
    int const    nIJ = nI * nJ;
    int const    nS = ks.extent(1);
    int const    nT = ks.extent(2);

    int const ik = ijk / nIJ;
    int const ij = ijk % nIJ / nI;
    int const ii = ijk % nIJ % nI;

    CuCxF        temp[NP] = {CuCxF(0.f),};
    float3 const r{(ii - nI / 2.f) / (float)nI, (ij - nJ / 2.f) / (float)nJ, (ik - nK / 2.f) / (float)nK};
    for (int it = 0; it < nT; it++) {
      for (int is = 0; is < nS; is++) {
        float3 const k = traj(is, it);
        float const  p = pi2 * (r.x * k.x + r.y * k.y + r.z * k.z);
        CuCxF const  ep(scale * cuda::std::cos(p), scale * cuda::std::sin(p));
        for (int ic = 0; ic < NP; ic++) {
          temp[ic] += ep * ks(ic, is, it);
        }
      }
    }
    for (int ic = 0; ic < NP; ic++) {
      imgs(ic, ii, ij, ik) = temp[ic];
    }
  });
  rl::Log::Print("DFT", "Adjoint Packed DFT finished in {}", rl::Log::ToNow(start));
}

template struct ThreeDPacked<8>;

} // namespace gw::DFT
